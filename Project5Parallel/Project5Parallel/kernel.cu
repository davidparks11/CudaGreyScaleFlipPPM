#include "hip/hip_runtime.h"
/******************************
*STUDENT NAME: DAVID PARKS    *
*PROJECT: 6 - GREY SCALE FLIP *
*DUE DATE: THURS 18/10/18     *
*******************************/
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include<stdio.h>
#include <iostream>
#include <fstream>
#include <stdexcept>
#include <sstream>

#define PPM_MAGIC_1 'P'
#define PPM_MAGIC_2 '6' 
#define BLOCK_SIZE 16;

struct PPM_header {
	int width;
	int height;
	int max_color;
};
struct RGB_8 {
	uint8_t r;
	uint8_t g;
	uint8_t b;
};//__attribute__((packed));

void PPM_read_header(std::ifstream &inp, PPM_header &ppm_header) {
	char ppm_magic_1, ppm_magic_2;
	inp >> ppm_magic_1;
	inp >> ppm_magic_2;

	if (ppm_magic_1 != PPM_MAGIC_1 || ppm_magic_2 != PPM_MAGIC_2) {
		throw std::runtime_error("File does not begin with PPM magic number");
	}

	int width;
	inp >> width;
	ppm_header.width = width;
	int height;
	inp >> height;
	ppm_header.height = height;

	int max_color;
	inp >> max_color;
	ppm_header.max_color = max_color;

	char space;
	//inp >> space;		// finish the header
	inp.read(&space, 1);

	return;
}

void PPM_read_rgb_8(std::ifstream &inp, int width, int height, RGB_8 *img) {
	inp.read((char *)img, sizeof(RGB_8)*width*height);
	if (!inp) {
		std::stringstream ss;
		ss << "error: only " << inp.gcount() << " could be read";
		throw std::runtime_error(ss.str());
	}
}

void PPM_write_header_8(std::ofstream &outp, int width, int height) {
	// write the header
	outp << PPM_MAGIC_1 << PPM_MAGIC_2 << (char)10 << width << (char)10
		<< height << (char)10 << 255 << (char)10;
}

void PPM_write_rgb_8(std::ofstream &outp, int width, int height, RGB_8 *img) {
	// write the image
	outp.write((char *)img, sizeof(RGB_8) * width * height);
	if (!outp) {
		std::stringstream ss;
		ss << "error: only " << outp.tellp() << " could be written";
		throw std::runtime_error(ss.str());
	}
}

////////////////////// STUDENT CODE (1/2) //////////////////////


__global__ void gray_scale_flip(RGB_8* img, int height, int width)
{
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;

	if (row < height && col < width / 2)
	{
		int i = row * width + col;
		//temp var for slip pixel
		RGB_8 temp = img[(row + 1) * width - col - 1];

		//computing gray value
		float gray_value = 0.21 * img[i].r + 0.72 * img[i].g + 0.07 * img[i].b;
		img[i].r = gray_value;
		img[i].g = gray_value;
		img[i].b = gray_value;

		//set flip pixel to grayed current pixel
		img[(row + 1) * width - col - 1] = img[i];

		//set current pixel to temp pixel
		img[i] = temp;

		//computing gray value
		gray_value = 0.21 * img[i].r + 0.72 * img[i].g + 0.07 * img[i].b;
		img[i].r = gray_value;
		img[i].g = gray_value;
		img[i].b = gray_value;
	}
}

////////////////////// END STUDENT CODE (1/2) //////////////////////


int main(int argc, char *argv[]) {
	if (argc != 3) {
		std::cerr << "Usage: " << argv[0] << " in_ppm_file out_ppm_file" << std::endl;
		return 1;
	}

	PPM_header img_header;
	hipError_t cudaStatus;


	try {
		std::ifstream ifs(argv[1], std::ios::binary);
		if (!ifs) {
			throw std::runtime_error("Cannot open input file");
		}

		PPM_read_header(ifs, img_header);
		std::cout << img_header.width << " " << img_header.height << " " << img_header.max_color << std::endl;

		RGB_8 *img = new RGB_8[img_header.height * img_header.width];

		PPM_read_rgb_8(ifs, img_header.width, img_header.height, (RGB_8 *)img);

		std::ofstream ofs(argv[2], std::ios::binary);
		if (!ofs) {
			throw std::runtime_error("Cannot open output file");
		}

		PPM_write_header_8(ofs, img_header.width, img_header.height);


		////////////////////// STUDENT CODE (2/2) //////////////////////
		RGB_8* img_device;
		int size = img_header.height * img_header.width;

		//allocate memory on GPU
		cudaStatus = hipMalloc((void**)&img_device, size * sizeof(RGB_8));
		if (cudaStatus != hipSuccess)
		{
			std::cout << "Could not allocate space on GPU" << std::endl;
			hipFree(img_device);
			return -1;
		}

		//copy memory from host to GPU
		cudaStatus = hipMemcpy(img_device, img, size * sizeof(RGB_8), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			std::cout << "Could not copy memory from host to GPU" << std::endl;
			hipFree(img_device);
			return -2;
		}

		//define block
		dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

		//define grid
		dim3 dim_grid(ceil(((float)img_header.width / 2) / BLOCK_SIZE), ceil((float)img_header.height / BLOCK_SIZE), 1);

		//call kernel function
		gray_scale_flip << <dim_grid, dim_block >> >(img_device, img_header.height, img_header.width);

		RGB_8 *new_img = (RGB_8*)malloc(sizeof(RGB_8) * img_header.height * img_header.width);

		//copy memory from host to GPU
		cudaStatus = hipMemcpy(new_img, img_device, size * sizeof(RGB_8), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			std::cout << "Could not copy memory from GPU to host" << std::endl;
			hipFree(img_device);
			return -3;
		}

		////////////////////// END STUDENT CODE (2/2) //////////////////////


		PPM_write_rgb_8(ofs, img_header.width, img_header.height, (RGB_8 *)new_img);

		hipFree(img_device);

		ifs.close();
		ofs.close();
	}
	catch (std::runtime_error &re) {
		std::cout << re.what() << std::endl;
		return 2;
	}

	return 0;
}
